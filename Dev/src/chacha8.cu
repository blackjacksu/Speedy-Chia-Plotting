
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>

#define U32TO32_LITTLE(v) (v)
#define U8TO32_LITTLE(p) (*(const uint32_t *)(p))
#define U32TO8_LITTLE(p, v) (((uint32_t *)(p))[0] = U32TO32_LITTLE(v))
#define ROTL32(v, n) (((v) << (n)) | ((v) >> (32 - (n))))

#define ROTATE(v, c) (ROTL32(v, c))
#define XOR(v, w) ((v) ^ (w))
#define PLUS(v, w) ((v) + (w))
#define PLUSONE(v) (PLUS((v), 1))

#define QUARTERROUND(a, b, c, d) \
    a = PLUS(a, b);              \
    d = ROTATE(XOR(d, a), 16);   \
    c = PLUS(c, d);              \
    b = ROTATE(XOR(b, c), 12);   \
    a = PLUS(a, b);              \
    d = ROTATE(XOR(d, a), 8);    \
    c = PLUS(c, d);              \
    b = ROTATE(XOR(b, c), 7)

struct chacha8_ctx {
    uint32_t input[16];
};

__global__ void chacha8_keysetup(struct chacha8_ctx *x, const uint8_t *k, uint32_t kbits, const uint8_t *iv)
{
    const char *constants;
    static const char sigma[17] = "expand 32-byte k";
    static const char tau[17] = "expand 16-byte k";
    
    x->input[4] = U8TO32_LITTLE(k + 0);
    x->input[5] = U8TO32_LITTLE(k + 4);
    x->input[6] = U8TO32_LITTLE(k + 8);
    x->input[7] = U8TO32_LITTLE(k + 12);
    if (kbits == 256) { /* recommended */
        k += 16;
        constants = sigma;
    } else { /* kbits == 128 */
        constants = tau;
    }
    x->input[8] = U8TO32_LITTLE(k + 0);
    x->input[9] = U8TO32_LITTLE(k + 4);
    x->input[10] = U8TO32_LITTLE(k + 8);
    x->input[11] = U8TO32_LITTLE(k + 12);
    x->input[0] = U8TO32_LITTLE(constants + 0);
    x->input[1] = U8TO32_LITTLE(constants + 4);
    x->input[2] = U8TO32_LITTLE(constants + 8);
    x->input[3] = U8TO32_LITTLE(constants + 12);
    if (iv) {
        x->input[14] = U8TO32_LITTLE(iv + 0);
        x->input[15] = U8TO32_LITTLE(iv + 4);
    } else {
        x->input[14] = 0;
        x->input[15] = 0;
    }
}

__global__ void chacha8_get_keystream(const struct chacha8_ctx *x, uint64_t pos, uint32_t n_blocks, uint8_t *c)
{
    uint32_t x0, x1, x2, x3, x4, x5, x6, x7, x8, x9, x10, x11, x12, x13, x14, x15;
    uint32_t j0, j1, j2, j3, j4, j5, j6, j7, j8, j9, j10, j11, j12, j13, j14, j15;
    int i;

    j0 = x->input[0];
    j1 = x->input[1];
    j2 = x->input[2];
    j3 = x->input[3];
    j4 = x->input[4];
    j5 = x->input[5];
    j6 = x->input[6];
    j7 = x->input[7];
    j8 = x->input[8];
    j9 = x->input[9];
    j10 = x->input[10];
    j11 = x->input[11];
    j12 = pos;
    j13 = pos >> 32;
    j14 = x->input[14];
    j15 = x->input[15];

    while (n_blocks--) {
        x0 = j0;
        x1 = j1;
        x2 = j2;
        x3 = j3;
        x4 = j4;
        x5 = j5;
        x6 = j6;
        x7 = j7;
        x8 = j8;
        x9 = j9;
        x10 = j10;
        x11 = j11;
        x12 = j12;
        x13 = j13;
        x14 = j14;
        x15 = j15;
        for (i = 8; i > 0; i -= 2) {
            QUARTERROUND(x0, x4, x8, x12);
            QUARTERROUND(x1, x5, x9, x13);
            QUARTERROUND(x2, x6, x10, x14);
            QUARTERROUND(x3, x7, x11, x15);
            QUARTERROUND(x0, x5, x10, x15);
            QUARTERROUND(x1, x6, x11, x12);
            QUARTERROUND(x2, x7, x8, x13);
            QUARTERROUND(x3, x4, x9, x14);
        }
        x0 = PLUS(x0, j0);
        x1 = PLUS(x1, j1);
        x2 = PLUS(x2, j2);
        x3 = PLUS(x3, j3);
        x4 = PLUS(x4, j4);
        x5 = PLUS(x5, j5);
        x6 = PLUS(x6, j6);
        x7 = PLUS(x7, j7);
        x8 = PLUS(x8, j8);
        x9 = PLUS(x9, j9);
        x10 = PLUS(x10, j10);
        x11 = PLUS(x11, j11);
        x12 = PLUS(x12, j12);
        x13 = PLUS(x13, j13);
        x14 = PLUS(x14, j14);
        x15 = PLUS(x15, j15);

        j12 = PLUSONE(j12);
        if (!j12) {
            j13 = PLUSONE(j13);
            /* stopping at 2^70 bytes per nonce is user's responsibility */
        }

        U32TO8_LITTLE(c + 0, x0);
        U32TO8_LITTLE(c + 4, x1);
        U32TO8_LITTLE(c + 8, x2);
        U32TO8_LITTLE(c + 12, x3);
        U32TO8_LITTLE(c + 16, x4);
        U32TO8_LITTLE(c + 20, x5);
        U32TO8_LITTLE(c + 24, x6);
        U32TO8_LITTLE(c + 28, x7);
        U32TO8_LITTLE(c + 32, x8);
        U32TO8_LITTLE(c + 36, x9);
        U32TO8_LITTLE(c + 40, x10);
        U32TO8_LITTLE(c + 44, x11);
        U32TO8_LITTLE(c + 48, x12);
        U32TO8_LITTLE(c + 52, x13);
        U32TO8_LITTLE(c + 56, x14);
        U32TO8_LITTLE(c + 60, x15);

        c += 64;
    }
}

extern "C" void perform_get_keystream(const struct chacha8_ctx *_x, uint64_t _pos, uint32_t _n_blocks, uint8_t *_c, const uint8_t N)
{
    // Device memory 

    struct chacha8_ctx *x;    
    uint64_t pos;
    uint32_t n_blocks;
    uint8_t c[64];

    hipEvent_t start, stop;
    float       elapsedTime;

    /* begin timing */
    hipEventCreate(&start);
    hipEventRecord(start, 0);

    //cudaMalloc(x, sizeof(chacha8_ctx));
    //cudaMalloc(pos, sizeof(uint64_t));
    //cudaMalloc(n_blocks, sizeof(uint32_t));
    // cudaMalloc(&c, sizeof(uint8_t));

    hipMemcpy(x, _x, sizeof(chacha8_ctx), hipMemcpyHostToDevice);
    hipMemcpy(&pos, &_pos, sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(&n_blocks, &_n_blocks, sizeof(uint32_t), hipMemcpyHostToDevice);

    // dim3 threadsPerBlock(8, 8, 8);
    // dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y, N / threadsPerBlock.z);
    int threadsPerBlock = 1;
    int numBlocks = 1;


    chacha8_get_keystream <<<numBlocks, threadsPerBlock>>>(x, pos, n_blocks, c);
    hipMemcpy(_c, c, sizeof(uint8_t) * 64, hipMemcpyDeviceToHost);
    /* end timing */
    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Execution time: %f seconds\n", elapsedTime / 1000);
    hipFree(&x);
    hipFree(&pos);
}
